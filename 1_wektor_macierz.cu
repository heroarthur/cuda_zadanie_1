
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h> 
#include <iostream> 
#include <stdio.h>


#define BLOCK_SIZE 16
using namespace std; 


typedef struct {
    int width;
    int height;
    int stride; 
    int* elements;
} Matrix;


typedef struct {
    int width;
    int* elements;
} Vector;
 

__device__ float GetElement(const Matrix A, int row, int col)
{
    return A.elements[row * A.stride + col];
}

__device__ void SetElement(Matrix A, int row, int col, float value)
{
    A.elements[row * A.stride + col] = value;
}



void print_matrix(const Matrix A) {
	int i;
	int size = A.width * A.height;
    cout<<"size A "<<size<<endl;
    cout<<"    MATRIX    \n";
	for(i = 1; i <= size; i++) {
		cout<<A.elements[i-1]<<" ";
		if(i % 10 == 0) {
			cout<<"\n";
		}
	}	
}


__global__ void macierz_wektor_10_kernel(const Matrix, const Vector, Vector);
void macierz_wektor_10()
{
    //create Matrix and Vector on Host (CPU)
    Matrix A;
    Vector B;
    Vector C;
    
    A.width = A.height = A.stride = 10;
    size_t size_A = A.width * A.height * sizeof(int);
    A.elements = (int*) malloc(size_A);

    B.width = 10;
    size_t size_B = B.width * sizeof(int);
    B.elements = (int*) malloc(size_B);

    C.width = 10;
    size_t size_C = C.width * sizeof(int);
    C.elements = (int*) malloc(size_C);

    int i;
    for(i = 0; i < A.width*A.height; i++) {
        A.elements[i] = (i % 10) + 1;
    }
	
    print_matrix(A);    

    for(i = 0; i < B.width; i++) {
        B.elements[i] = (i % 10) + 1;
    }    

    //Load A and B to device memory
    Matrix d_A;
    d_A.width = d_A.stride = A.width; d_A.height = A.height;
    hipMalloc(&d_A.elements, size_A);
    hipMemcpy(d_A.elements, A.elements, size_A, hipMemcpyHostToDevice);

    Vector d_B;
    d_B.width = B.width;
    hipMalloc(&d_B.elements, size_B);
    hipMemcpy(d_B.elements, B.elements, size_B, hipMemcpyHostToDevice);

    Vector d_C;
    d_C.width = C.width;
    hipMalloc(&d_C.elements, size_C);

    
    dim3 dimBlock(10, 1);
    dim3 dimGrid(1);
    macierz_wektor_10_kernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

    hipMemcpy(C.elements, d_C.elements, size_C, hipMemcpyDeviceToHost);
    for(i = 0; i < C.width; i++) {
        cout<<C.elements[i]<<" ";
    }
    cout<<endl;    
}


__global__ void macierz_wektor_10_kernel(const Matrix A, const Vector B, Vector C) {
    int col = threadIdx.x;
    printf("thread_id_x %d", threadIdx.x);
    int vec_val = B.elements[col];
    int mul = 0;
    int row;
    for(row = 0; row < A.height; row++) {
        mul += vec_val * A.elements[row*A.width + col];
    }
    C.elements[col] = mul;
    __syncthreads();
}




int main()
{ 
    macierz_wektor_10();    

    return 0;
}

